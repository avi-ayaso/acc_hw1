#include "hip/hip_runtime.h"
#include "ex1.h"

#define VEC_SIZE 256
#define HISTOGRAM_SIZE 256




__device__ void prefix_sum(int *arr , int len) {
    
    int thIdx = threadIdx.x;
    int inc;
    
    for(int stride = 1 ; stride < len ; stride *= 2){
                
        if(thIdx >= stride &&  thIdx < len){
            inc = arr[thIdx - stride];
        }    
        __syncthreads();
        if(thIdx >= stride && thIdx < len){
            arr[thIdx] += inc;
        }
        __syncthreads();
    }
    return; 
}

__device__ void map_calc(uchar *map ,int *cdf, int idx){
    float map_value = IMG_HEIGHT * IMG_WIDTH;
    if(idx < HISTOGRAM_SIZE){
        map[idx] = ((uchar)(N_COLORS * (cdf[idx] /map_value))) * (256 / N_COLORS);
    }
}

__global__ void process_image_kernel(uchar *all_in, uchar *all_out){
    int thIdx = threadIdx.x;
    int offset = IMG_WIDTH * IMG_HEIGHT * blockIdx.x + thIdx;
    __shared__ int histogram[HISTOGRAM_SIZE];
    __shared__ uchar map[HISTOGRAM_SIZE];
    int * cdf = histogram;
    if (thIdx < HISTOGRAM_SIZE) {
        histogram[thIdx] = 0;
    }
    __syncthreads();
    for(int j = 0; j < IMG_WIDTH * IMG_HEIGHT; j += blockDim.x){
        int pixelValue = all_in[offset + j];
        atomicAdd(histogram + pixelValue, 1);
    }
    __syncthreads();
    prefix_sum(histogram, HISTOGRAM_SIZE);
    map_calc(map ,cdf,thIdx);
    __syncthreads();
    for(int j = 0; j < IMG_WIDTH * IMG_HEIGHT; j += blockDim.x){
        int pixelValue = all_in[offset + j];
        all_out[offset + j] = map[pixelValue];
    }
    return;
}


/* Task serial context struct with necessary CPU / GPU pointers to process a single image */
struct task_serial_context {
    // TODO define task serial memory buffers
    uchar *gpu_in_img[N_IMAGES];
    uchar *gpu_out_img[N_IMAGES];
};

/* Allocate GPU memory for a single input image and a single output image.
 * 
 * Returns: allocated and initialized task_serial_context. */
struct task_serial_context *task_serial_init()
{
    auto context = new task_serial_context;
    //allocate GPU memory for a single input image and a single output image
    for(int i = 0 ; i < N_IMAGES ; i++){
        CUDA_CHECK( hipMalloc(&context->gpu_in_img[i], IMG_HEIGHT * IMG_WIDTH) );
        CUDA_CHECK( hipMalloc(&context->gpu_out_img[i], IMG_HEIGHT * IMG_WIDTH) );
    }
    return context;
}

/* Process all the images in the given host array and return the output in the
 * provided output host array */
void task_serial_process(struct task_serial_context *context, uchar *images_in, uchar *images_out)
{
    
    //TODO: in a for loop:
    int offset = 0;
    for(int i = 0 ; i < N_IMAGES ; i++ , offset += IMG_HEIGHT * IMG_WIDTH){
        //   1. copy the relevant image from images_in to the GPU memory you allocated
        // offset = i * IMG_HEIGHT * IMG_WIDTH ;
        CUDA_CHECK( hipMemcpy(context->gpu_in_img[i] , images_in + offset , IMG_HEIGHT * IMG_WIDTH, hipMemcpyHostToDevice) );
        //   2. invoke GPU kernel on this image  
        process_image_kernel<<<1 , 1024>>>(context->gpu_in_img[i] , context->gpu_out_img[i]);
        //   3. copy output from GPU memory to relevant location in images_out_gpu_serial
        CUDA_CHECK( hipMemcpy(images_out + offset , context->gpu_out_img[i] , IMG_HEIGHT * IMG_WIDTH, hipMemcpyDeviceToHost) );
    }
    CUDA_CHECK( hipDeviceSynchronize() );
    
}

/* Release allocated resources for the task-serial implementation. */
void task_serial_free(struct task_serial_context *context)
{
    //TODO: free resources allocated in task_serial_init
    for(int i = 0 ; i < N_IMAGES ; i++){
        CUDA_CHECK( hipFree(context->gpu_in_img[i]) );
        CUDA_CHECK( hipFree(context->gpu_out_img[i]) );
    }
    free(context);
}

/* Bulk GPU context struct with necessary CPU / GPU pointers to process all the images */
struct gpu_bulk_context {
    // TODO define bulk-GPU memory buffers
    uchar *gpu_in_imgs;
    uchar *gpu_out_imgs;
};

/* Allocate GPU memory for all the input and output images.
 * 
 * Returns: allocated and initialized gpu_bulk_context. */
struct gpu_bulk_context *gpu_bulk_init()
{
    auto context = new gpu_bulk_context;

    //TODO: allocate GPU memory for a all input images and all output images
    CUDA_CHECK( hipMalloc(&context->gpu_in_imgs, N_IMAGES * IMG_HEIGHT * IMG_WIDTH) );
    CUDA_CHECK( hipMalloc(&context->gpu_out_imgs, N_IMAGES * IMG_HEIGHT * IMG_WIDTH) );
    return context;
}

/* Process all the images in the given host array and return the output in the
 * provided output host array */
void gpu_bulk_process(struct gpu_bulk_context *context, uchar *images_in, uchar *images_out)
{
    //   1. copy all input images from images_in to the GPU memory you allocated
    CUDA_CHECK( hipMemcpy(context->gpu_in_imgs , images_in, N_IMAGES * IMG_HEIGHT * IMG_WIDTH, hipMemcpyHostToDevice) );

    //   2. invoke a kernel with N_IMAGES threadblocks, each working on a different image
    process_image_kernel<<<N_IMAGES , 1024>>>(context->gpu_in_imgs , context->gpu_out_imgs);

    //   3. copy output images from GPU memory to images_out
    CUDA_CHECK( hipMemcpy(images_out, context->gpu_out_imgs , N_IMAGES * IMG_HEIGHT * IMG_WIDTH, hipMemcpyDeviceToHost) );

}

/* Release allocated resources for the bulk GPU implementation. */
void gpu_bulk_free(struct gpu_bulk_context *context)
{
    //free resources allocated in gpu_bulk_init
    CUDA_CHECK(hipFree(context->gpu_in_imgs));
    CUDA_CHECK(hipFree(context->gpu_out_imgs));
    free(context);

}
